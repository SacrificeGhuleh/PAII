#include "CudaTimer.cuh"
#include <cudaDefs.h>

namespace nscommon {
  CudaTimer::CudaTimer() : elapsed_{0}, running_{false} {
    hipEventCreate(&startEvent_);
    hipEventCreate(&stopEvent_);
  }

  CudaTimer::~CudaTimer() {
    hipEventDestroy(startEvent_);
    hipEventDestroy(stopEvent_);
  }

  void CudaTimer::start(hipStream_t stream) {
    if (running_) {
      throw std::runtime_error("Timer is already running");
    }
    running_ = true;
    hipEventRecord(startEvent_, stream);
  }

  float CudaTimer::stop(hipStream_t stream) {
    if (!running_) {
      throw std::runtime_error("Timer is not running");
    }
    running_ = false;
    hipEventRecord(stopEvent_, stream);
    hipEventSynchronize(stopEvent_);
    hipEventElapsedTime(&elapsed_, startEvent_, stopEvent_);
    return elapsed_;
  }
}
